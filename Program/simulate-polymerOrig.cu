//Libraries


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

//Defines

#define M_PI 3.14159265358979323846

#define h 0.001

#define m 1.0

#define k_e 100.0
#define b_pol 1.0
#define k_b 0.0

#define sig 1.5
#define eps 3.0
#define cutoff pow(2,1.0/6.0)*sig

//Structs

struct sim_params
{
  float T;
  float eta;
  int N;
  float f_pull;
  char IC_type;
  float meas_freq;
  float sim_time;
};

typedef struct sim_params sim_params;
typedef struct hiprandStatePhilox4_32_10 PRNGstate;

//Functions

void read_parameters( struct sim_params *sp, FILE *f)
{
  fscanf(f,"T\t%f\n",&(sp->T));
  fscanf(f,"eta\t%f\n",&(sp->eta));
  fscanf(f,"N\t%d\n",&(sp->N));
  fscanf(f,"f_pull\t%f\n",&(sp->f_pull));
  fscanf(f,"IC_type\t%c\n",&(sp->IC_type));
  fscanf(f,"meas_freq\t%f\n",&(sp->meas_freq));
  fscanf(f,"sim_time\t%f\n",&(sp->sim_time));
}

void write_gro_frame( int N, float *r, FILE *f)
{
  fprintf(f,"MD simulation of a polymer, t = 0.0\n");
  fprintf(f,"%5d\n",N);
  for( int i_p=0; i_p<N; i_p++)
  {
    fprintf(f,"%5d%-5s%5s%5d%8.3f%8.3f%8.3f\n",i_p+1,"X","X",i_p+1,r[3*i_p+0],r[3*i_p+1],r[3*i_p+2]);
  }
  fprintf(f,"%10.5f%10.5f%10.5f\n",0.0,0.0,0.0);
}

void write_trr_frame( int N, float *r, int i_f, float t, FILE *f)
{
  //header
  int magickvalue=1993;
  fwrite(&magickvalue,sizeof(int),1,f);
  char trrversion[]="GMX_trn_file";
  int len_s_a=sizeof(trrversion);
  int len_s_b=sizeof(trrversion)-1;
  fwrite(&len_s_a,sizeof(int),1,f);
  fwrite(&len_s_b,sizeof(int),1,f);
  fwrite(trrversion,sizeof(char),sizeof(trrversion)-1,f);
  int zero=0;
  for( int i=0; i<7; i++)
  {
    fwrite(&zero,sizeof(int),1,f);
  }
  int x_size=3*N*sizeof(float);
  fwrite(&x_size,sizeof(int),1,f);
  int v_size=0;
  fwrite(&v_size,sizeof(int),1,f);
  int f_size=0;
  fwrite(&f_size,sizeof(int),1,f);
  int natoms=N;
  fwrite(&natoms,sizeof(int),1,f);
  int step=i_f;
  fwrite(&step,sizeof(int),1,f);
  int time=t;
  fwrite(&zero,sizeof(int),1,f);
  fwrite(&time,sizeof(float),1,f);
  fwrite(&zero,sizeof(int),1,f);
  //coordinates
  fwrite(r,sizeof(float),3*N,f);
}

void write_checkpoint( int N, float *r, float *v, float t, int n_threads, PRNGstate *state, int f_idx, FILE *f)
{
  int natoms=N;
  fwrite(&natoms,sizeof(int),1,f);
  int time=t;
  fwrite(&time,sizeof(float),1,f);
  int index=f_idx;
  fwrite(&index,sizeof(int),1,f);
  fwrite(r,sizeof(float),3*N,f);
  fwrite(v,sizeof(float),3*N,f);
  fwrite(state,sizeof(PRNGstate),n_threads,f);
}

void read_checkpoint( int N, float *r, float *v, float *t, int n_threads, PRNGstate *state, int *f_idx, FILE *f)
{
  int natoms;
  fread(&natoms,sizeof(int),1,f);
  if( natoms!=N){ printf("Error reading checkpoint.\n"); exit(-1);}
  int time;
  fread(&time,sizeof(float),1,f);
  int index;
  fread(&index,sizeof(int),1,f);
  fread(r,sizeof(float),3*N,f);
  fread(v,sizeof(float),3*N,f);
  fread(state,sizeof(PRNGstate),n_threads,f);
  *t=time;
  *f_idx=index+1;
}

void set_random_IC( int N, float T, float *r, float *v)
{
  hiprandGenerator_t gen;
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL));
  float random, theta, varphi, bondlen, bondangle;
  float dir_old[3], dir_new[3], perpdir[3], perpdirnorm;
  hiprandGenerateUniform(gen,&random,1); theta = acos(1.0-2.0*random); 
  hiprandGenerateUniform(gen,&random,1); varphi = 2.0*M_PI*random;
  dir_old[0]=sin(theta)*cos(varphi);
  dir_old[1]=sin(theta)*sin(varphi);
  dir_old[2]=cos(theta);
  r[0]=r[1]=r[2]=0.0;
  for( int i_p=1; i_p<N; i_p++)
  {
    hiprandGenerateUniform(gen,&random,1); theta = acos(1.0-2.0*random); 
    hiprandGenerateUniform(gen,&random,1); varphi = 2.0*M_PI*random;
    perpdir[0] = dir_old[1]*cos(theta)-dir_old[2]*sin(theta)*sin(varphi);
    perpdir[1] = dir_old[2]*sin(theta)*cos(varphi)-dir_old[0]*cos(theta);
    perpdir[2] = dir_old[0]*sin(theta)*sin(varphi)-dir_old[1]*sin(theta)*cos(varphi);
    perpdirnorm = sqrt(perpdir[0]*perpdir[0]+perpdir[1]*perpdir[1]+perpdir[2]*perpdir[2]);
    perpdir[0] /= perpdirnorm; perpdir[1] /= perpdirnorm; perpdir[2] /= perpdirnorm;
    hiprandGenerateUniform(gen,&random,1);
    if( k_b<__FLT_MIN__){ bondangle = acos(1.0-2.0*random);}
    else{ bondangle = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b/T)))*random)/(k_b/T));}
    dir_new[0] = dir_old[0]*cos(bondangle)+perpdir[0]*sin(bondangle);
    dir_new[1] = dir_old[1]*cos(bondangle)+perpdir[1]*sin(bondangle);
    dir_new[2] = dir_old[2]*cos(bondangle)+perpdir[2]*sin(bondangle);
    hiprandGenerateUniform(gen,&random,1);
    bondlen = b_pol+sqrt(2.0)*sqrt(T/k_e)*erfinv(2.0*random-1.0);
    r[3*i_p+0] = bondlen*dir_new[0]+r[3*(i_p-1)+0];
    r[3*i_p+1] = bondlen*dir_new[1]+r[3*(i_p-1)+1];
    r[3*i_p+2] = bondlen*dir_new[2]+r[3*(i_p-1)+2];
    int overlap = 0;
    for(int j_p=0; j_p<(i_p-1); j_p++)
    {
      float dist = 0.0;
      dist += (r[3*j_p+0]-r[3*i_p+0])*(r[3*j_p+0]-r[3*i_p+0]);
      dist += (r[3*j_p+1]-r[3*i_p+1])*(r[3*j_p+1]-r[3*i_p+1]);
      dist += (r[3*j_p+2]-r[3*i_p+2])*(r[3*j_p+2]-r[3*i_p+2]);
      dist = sqrt(dist);
      if( dist<cutoff){ overlap=1; break;}
    }
    if( overlap==0)
    {
      dir_old[0] = dir_new[0];
      dir_old[1] = dir_new[1];
      dir_old[2] = dir_new[2];
    }
    else{ i_p--;}
  }
  for( int i_p=0; i_p<N; i_p++)
  {
    for( int i_c=0; i_c<3; i_c++)
    {
      hiprandGenerateUniform(gen,&random,1);
      v[3*i_p+i_c] = sqrt(2.0)*sqrt(T/m)*erfinv(2.0*random-1.0);
    }
  }
  hiprandDestroyGenerator(gen);
}

void set_linear_IC( int N, float *r, float *v)
{
  r[0]=r[1]=r[2]=0.0;
  for( int i_p=1; i_p<N; i_p++)
  {
    r[3*i_p+0] = b_pol+r[3*(i_p-1)+0];
    r[3*i_p+1] = r[3*(i_p-1)+1];
    r[3*i_p+2] = r[3*(i_p-1)+2];
  }
  for( int i_p=0; i_p<N; i_p++)
  {
    for( int i_c=0; i_c<3; i_c++)
    {
      v[3*i_p+i_c] = 0.0;
    }
  }
}

//Kernels

__global__
void setup_PRNG( int seed, PRNGstate *state)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, i_p, 0, &state[i_p]);
}

__global__
void call_PRNG( float c_rn, float *nrn, PRNGstate *state)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  for( int i_c = 0; i_c < 3; i_c++)
  {
    nrn[3*i_p+i_c] = c_rn*hiprand_normal(&state[i_p]);
  }
}

__global__
void calc_extern_f( int N, float eta, float *v, float *f_c, float *f)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N)
  {
    for( int i_c = 0; i_c < 3; i_c++)
    {
      f[3*i_p+i_c] = f_c[3*i_p+i_c];

      f[3*i_p+i_c] += -eta*v[3*i_p+i_c];
    }
  }
}

__global__ 
void calc_bonds( int N, float *r, float *b, float *invlen)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N-1)
  {
    invlen[i_p+2] = 0.0;
    for( int i_c = 0; i_c < 3; i_c++)
    {
      b[3*(i_p+2)+i_c] = r[3*(i_p+1)+i_c]-r[3*i_p+i_c];
      invlen[i_p+2] += b[3*(i_p+2)+i_c]*b[3*(i_p+2)+i_c];
    }
    invlen[i_p+2] = 1.0/sqrt(invlen[i_p+2]);
  }
}

__global__
void calc_cosines( int N, float *b, float *invlen, float *cosine)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N-2)
  {
    cosine[i_p+3] = 0.0;
    for( int i_c = 0; i_c < 3; i_c++)
    {
      cosine[i_p+3] += b[3*(i_p+3)+i_c]*b[3*(i_p+2)+i_c];
    }
    cosine[i_p+3] *= invlen[i_p+3]*invlen[i_p+2];
  }
}

__global__
void calc_intern_f( int N, float *b, float *invlen, float *cosine, float *f)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N)
  {
    for( int i_c = 0; i_c < 3; i_c++)
    {
      f[3*i_p+i_c] += k_e*(1.0-b_pol*invlen[i_p+1])*(-b[3*(i_p+1)+i_c]);

      f[3*i_p+i_c] += k_e*(1.0-b_pol*invlen[i_p+2])*(+b[3*(i_p+2)+i_c]);

      f[3*i_p+i_c] += k_b*(+b[3*(i_p+0)+i_c])*invlen[i_p+1]*invlen[i_p+0];

      f[3*i_p+i_c] += k_b*(+b[3*(i_p+2)+i_c]-b[3*(i_p+1)+i_c])*invlen[i_p+2]*invlen[i_p+1];

      f[3*i_p+i_c] += k_b*(-b[3*(i_p+3)+i_c])*invlen[i_p+3]*invlen[i_p+2];

      f[3*i_p+i_c] += k_b*(-cosine[i_p+2]-cosine[i_p+1])*b[3*(i_p+1)+i_c]*invlen[i_p+1]*invlen[i_p+1];

      f[3*i_p+i_c] += k_b*(+cosine[i_p+3]+cosine[i_p+2])*b[3*(i_p+2)+i_c]*invlen[i_p+2]*invlen[i_p+2];
    }
  }
}

__global__
void calc_exclvol_f( int N, float *r, float *f)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N)
  {
    int skip;
    float d2;
    float k_LJ;
    for( int j_p=i_p-2; j_p>=0; j_p-=1+skip)
    {
      d2 = 0.0;
      for( int i_c=0; i_c<3; i_c++)
      {
        d2 += (r[3*i_p+i_c]-r[3*j_p+i_c])*(r[3*i_p+i_c]-r[3*j_p+i_c]);
      }
      if( d2<pow(cutoff,2))
      {
        k_LJ = 4.0*eps*(12.0*pow(sig,12)/(d2*d2*d2*d2*d2*d2*d2)-6.0*pow(sig,6)/(d2*d2*d2*d2));
        for( int i_c = 0; i_c < 3; i_c++)
        {
          f[3*i_p+i_c] += k_LJ*(r[3*i_p+i_c]-r[3*j_p+i_c]);
        }
        skip=0;
      }
      else
      {
        skip=((sqrt(d2)-cutoff)/(1.5*b_pol));
      }
    }
    for( int j_p=i_p+2; j_p<N; j_p+=1+skip)
    {
      d2 = 0.0;
      for( int i_c=0; i_c<3; i_c++)
      {
        d2 += (r[3*i_p+i_c]-r[3*j_p+i_c])*(r[3*i_p+i_c]-r[3*j_p+i_c]);
      }
      if( d2<pow(cutoff,2))
      {
        k_LJ = 4.0*eps*(12.0*pow(sig,12)/(d2*d2*d2*d2*d2*d2*d2)-6.0*pow(sig,6)/(d2*d2*d2*d2));
        for( int i_c = 0; i_c < 3; i_c++)
        {
          f[3*i_p+i_c] += k_LJ*(r[3*i_p+i_c]-r[3*j_p+i_c]);
        }
        skip=0;
      }
      else
      {
        skip=((sqrt(d2)-cutoff)/(1.5*b_pol));
      }
    }
  }
}

__global__
void RK_stage_1( int N, float eta, float *r_1, float *r_2, float *v_1, float *v_2, float *f_1, float *nrn)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N)
  {
    for( int i_c = 0; i_c < 3; i_c++)
    {
      r_2[3*i_p+i_c] = r_1[3*i_p+i_c]+v_1[3*i_p+i_c]*h;

      v_2[3*i_p+i_c] = v_1[3*i_p+i_c]+f_1[3*i_p+i_c]*h/m+nrn[3*i_p+i_c]/m;
    }
  }
}

__global__
void RK_stage_2( int N, float eta, float *r_1, float *v_1, float *v_2, float *f_1, float *f_2, float *nrn)
{
  int i_p = blockIdx.x * blockDim.x + threadIdx.x;
  if( i_p<N)
  {
    for( int i_c = 0; i_c < 3; i_c++)
    {
      r_1[3*i_p+i_c] = r_1[3*i_p+i_c]+0.5*(v_1[3*i_p+i_c]+v_2[3*i_p+i_c])*h;

      v_1[3*i_p+i_c] = v_1[3*i_p+i_c]+0.5*(f_1[3*i_p+i_c]+f_2[3*i_p+i_c])*h/m+nrn[3*i_p+i_c]/m;
    }
  }
}

int main( int argc, char const *argv[])
{
  if( argc!=2){
    if( argc<2){ printf("You forgot the input.\n"); exit(-1);}
    else{ printf("Too many arguments.\n"); exit(-1);}
  }

  if( sizeof(argv[1])>128){ printf("Directory name too long.\n"); exit(-1);}
  char sim_dir[128];
  snprintf(sim_dir,sizeof(sim_dir),"%s",argv[1]);

  FILE *file_in;
  FILE *file_out;

  char filename[256];

  //Simulation parameters and variables

  sim_params sp;

  snprintf(filename,sizeof(filename),"%s/parameters.dat",sim_dir);
  file_in=fopen(filename,"rt");
  if( file_in==NULL){ printf("Error opening parameters file.\n"); exit(-1);}
  read_parameters(&sp,file_in);
  fclose(file_in);

  float *r_1;
  float *r_2;

  float *v_1;
  float *v_2;

  float *f_1;
  float *f_2;

  float *nrn;
  PRNGstate *state;

  float c_rn = sqrt(2*sp.eta*sp.T*h);

  float *b;
  float *invlen;
  float *cosine;

  float *f_c;

  size_t threads_block = 256;
  size_t n_blocks = (sp.N+threads_block-1)/threads_block;
  int n_threads = n_blocks*threads_block;

  //Memory allocation

  hipMallocManaged( &r_1, 3*sp.N*sizeof(float));
  hipMallocManaged( &r_2, 3*sp.N*sizeof(float));

  hipMallocManaged( &v_1, 3*sp.N*sizeof(float));
  hipMallocManaged( &v_2, 3*sp.N*sizeof(float));

  hipMallocManaged( &f_1, 3*sp.N*sizeof(float));
  hipMallocManaged( &f_2, 3*sp.N*sizeof(float));

  hipMallocManaged( &nrn, 3*n_threads*sizeof(float));
  hipMallocManaged( &state, n_threads*sizeof(PRNGstate));

  hipMallocManaged( &b, 3*(sp.N+3)*sizeof(float));
  hipMallocManaged( &invlen, (sp.N+3)*sizeof(float));
  hipMallocManaged( &cosine, (sp.N+4)*sizeof(float));

  hipMallocManaged( &f_c, 3*sp.N*sizeof(float));

  //Exceptions for the polymer ends

  b[0]=b[1]=b[2]=0.0;
  b[3]=b[4]=b[5]=0.0;
  b[3*(sp.N+2)+0]=b[3*(sp.N+2)+1]=b[3*(sp.N+2)+2]=0.0;
  b[3*(sp.N+1)+0]=b[3*(sp.N+1)+1]=b[3*(sp.N+1)+2]=0.0;

  invlen[0]=invlen[1]=0.0;
  invlen[sp.N+2]=invlen[sp.N+1]=0.0;

  cosine[0]=cosine[1]=cosine[2]=0.0;
  cosine[sp.N+3]=cosine[sp.N+2]=cosine[sp.N+1]=0.0;

  //Constant force

  for( int i_p=0; i_p<sp.N; i_p++)
  {
    for( int i_c=0; i_c<3; i_c++)
    {
      f_c[3*i_p+i_c]=0.0;
    }
  }
  f_c[3*0+0] = -sp.f_pull;
  f_c[3*(sp.N-1)+0] = sp.f_pull;

  //PRNG initialization

  setup_PRNG<<<n_blocks,threads_block>>>(time(NULL),state);
  hipDeviceSynchronize();

  //Initial condition

  float t = 0.0;

  int f_idx = 0;

  snprintf(filename,sizeof(filename),"%s/checkpoint.bin",sim_dir);
  file_in=fopen(filename,"rb");

  if( file_in==NULL)
  {
    if( sp.IC_type=='f')
    {
      snprintf(filename,sizeof(filename),"%s/initial-condition.bin",sim_dir);
      file_in=fopen(filename,"rb");
      if( file_in==NULL){ printf("Error opening file.\n"); exit(-1);}
      read_checkpoint(sp.N,r_1,v_1,&t,n_threads,state,&f_idx,file_in);
      fclose(file_in);
      t = 0.0;
      f_idx = 0;
    }
    else if( sp.IC_type=='r')
    {
      set_random_IC(sp.N,sp.T,r_1,v_1);
    }
    else if( sp.IC_type=='l')
    {
      set_linear_IC(sp.N,r_1,v_1);
    }
    else
    {
      printf("Unknown IC.\n");
      exit(-1);
    }
  }
  else
  {
    read_checkpoint(sp.N,r_1,v_1,&t,n_threads,state,&f_idx,file_in);
    fclose(file_in);
  }

  //Simulation

  printf("%s parameters: T=%06.3f eta=%06.3f N=%04d f_pull=%06.3f\n",sim_dir,sp.T,sp.eta,sp.N,sp.f_pull);
  printf("IC_type=%c meas_freq=%06.3f sim_time=%08.1f f_idx=%d\n",sp.IC_type,sp.meas_freq,sp.sim_time,f_idx);

  if( f_idx==0)
  {
    snprintf(filename,sizeof(filename),"%s/initial-condition.gro",sim_dir);
    file_out=fopen(filename,"wt");
    if( file_out==NULL){ printf("Error opening file.\n"); exit(-1);}
    write_gro_frame(sp.N,r_1,file_out);
    fclose(file_out);
  }

  snprintf(filename,sizeof(filename),"%s/trajectory-file-%d.trr",sim_dir,f_idx);
  file_out=fopen(filename,"wb");
  if( file_out==NULL){ printf("Error opening file.\n"); exit(-1);}

  int i_s;
  int n_steps = round((1.0/sp.meas_freq)/h);
  int i_f;
  int n_frames = round(sp.sim_time*sp.meas_freq);

  for( i_f = 0; i_f < n_frames; i_f++)
  {
    printf("Progress:%05.1lf%%\r",(100.0*i_f)/(1.0*n_frames));
    fflush(stdout);

    for( i_s = 0; i_s < n_steps; i_s++)
    {
      call_PRNG<<<n_blocks,threads_block>>>(c_rn,nrn,state);

      calc_extern_f<<<n_blocks,threads_block>>>(sp.N,sp.eta,v_1,f_c,f_1);
      calc_bonds<<<n_blocks,threads_block>>>(sp.N,r_1,b,invlen);
      calc_cosines<<<n_blocks,threads_block>>>(sp.N,b,invlen,cosine);
      calc_intern_f<<<n_blocks,threads_block>>>(sp.N,b,invlen,cosine,f_1);
      calc_exclvol_f<<<n_blocks,threads_block>>>(sp.N,r_1,f_1);

      RK_stage_1<<<n_blocks,threads_block>>>(sp.N,sp.eta,r_1,r_2,v_1,v_2,f_1,nrn);

      calc_extern_f<<<n_blocks,threads_block>>>(sp.N,sp.eta,v_2,f_c,f_2);
      calc_bonds<<<n_blocks,threads_block>>>(sp.N,r_2,b,invlen);
      calc_cosines<<<n_blocks,threads_block>>>(sp.N,b,invlen,cosine);
      calc_intern_f<<<n_blocks,threads_block>>>(sp.N,b,invlen,cosine,f_2);
      calc_exclvol_f<<<n_blocks,threads_block>>>(sp.N,r_2,f_2);

      RK_stage_2<<<n_blocks,threads_block>>>(sp.N,sp.eta,r_1,v_1,v_2,f_1,f_2,nrn);

      hipDeviceSynchronize();
    }

    t+=n_steps*h;

    write_trr_frame(sp.N,r_1,i_f,t,file_out);
  }

  fclose(file_out);

  snprintf(filename,sizeof(filename),"%s/checkpoint.bin",sim_dir);
  file_out=fopen(filename,"wb");
  if( file_out==NULL){ printf("Error opening file.\n"); exit(-1);}
  write_checkpoint(sp.N,r_1,v_1,t,n_threads,state,f_idx,file_out);
  fclose(file_out);

  //Memory deallocation

  hipFree(r_1);
  hipFree(r_2);

  hipFree(v_1);
  hipFree(v_2);

  hipFree(f_1);
  hipFree(f_2);

  hipFree(nrn);
  hipFree(state);

  hipFree(b);
  hipFree(invlen);
  hipFree(cosine);

  hipFree(f_c);
  
  return 0;
}
