/////////////////////////////////////////////////////////////////////
// PROGRAMA GENERADOR DE NUDOS DE PEDRO MENDIVIL Y CHAT GPT        //
// CALCULA VECTORES DE POSICION INICIALES DE CADENAS DE POLIMEROS  //
// SIRVE PARA ALIMENTAR AL PROGRAMA simulate-polimer.cu            //
// VERSION V1.1    noviembre 2024                               //
/////////////////////////////////////////////////////////////////////

//Libraries     

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> // Necesario para system()
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <string.h>

// Global variables
int subir = 1;
int bajar = 0;
int n_atom = 0;
int input_in_x = 1;
int input_in_y = 0;
int input_in_z = 0;
int input_in_n_atom = 0;
int atomo_validado =0;
float x=0;
float y=0;
float z=0;
int tecla = 0;
int tecla_ant = 0;

////// AQUI SE COPIAN PARTES DE TODA UNA SECCION DE simulate-polymer.cu ///////

// Variables globales  ///////////////////////////////////////////////
int no_atomo; // Antes N lo usaremos para ir rellenando el fichero de nudo
// npartic = número de partículas. Para random usamos global_N.
int npartic; // se modifica con cada xx.gro leido.
// file_copy[100] array global, copia de file_to_copy de void read_parameters()
char file_copy[100]; // cadena que contiene el nombre del fichero de nudo
// Todas las variables de parameters a global variables
float global_T;
float global_eta;
int global_N;
int global_N_random;
float global_f_pull;
char global_IC_type;
float global_meas_freq;
float global_sim_time;

// Structs   ///////////////////////////////////////////////////////

struct sim_params
{
  float T;           //          T   = Temperature
  float eta;         //         eta  = Viscosity
  int N;             //          N   = Number of particules (only for random)
  float f_pull;      //       f_pull = Pull force
  char IC_type;      //      IC_type = Initial condition type (f!!,r o l (load)) 
  // f do not use(checkpoint), r => chain random, l => load file XX.gro (knot)
  float meas_freq;   //    meas_freq = Frecuency of measurement (frames)
  float sim_time;    //     sim_time = Time of simulation
  char file_to_load[100]; // file_to_load = File format .gro to be read
};

typedef struct sim_params sim_params;
typedef struct hiprandStatePhilox4_32_10 PRNGstate;

//Functions   ///////////////////////////////////////////////////////////////

/////////////////// read_parameters & write global_vars  ////////////////////

void read_parameters( struct sim_params *sp, FILE *f)
{
  fscanf(f,"T\t%f\n",&(sp->T));
  fscanf(f,"eta\t%f\n",&(sp->eta));
  fscanf(f,"N\t%d\n",&(sp->N));
  fscanf(f,"f_pull\t%f\n",&(sp->f_pull));
  fscanf(f,"IC_type\t%c\n",&(sp->IC_type));
  fscanf(f,"meas_freq\t%f\n",&(sp->meas_freq));
  fscanf(f,"sim_time\t%f\n",&(sp->sim_time));
   
  if (fscanf(f, "file_to_load\t%99s\n", sp->file_to_load) != 1) {
      fprintf(stderr, "Error al leer file_to_load\n");
      exit(EXIT_FAILURE);
  }
  // Copiamos el contenido de file_to_load en file_copy que es global
  strcpy(file_copy, sp->file_to_load); // Copia el contenido de file_to_load en file_copy
  
  // Copiamos los demás valores de la estructura a las variables globales
  global_T = sp->T;
  global_eta = sp->eta;
  global_N = sp->N; // Redefinir inmediatamente con make_global_N
  global_f_pull = sp->f_pull;
  global_IC_type = sp->IC_type;
  global_meas_freq = sp->meas_freq;
  global_sim_time = sp->sim_time;
} 

//////////////////////////////// make_global_N ///////////////////////
int make_global_N() 
{
  FILE *knot = fopen(file_copy, "r");
  if (knot == NULL) {
     fprintf(stderr, "Error en la apertura del archivo de nudo\n");
     exit(EXIT_FAILURE);
  }
  char line[256];
  fgets(line, sizeof(line), knot); // Primera linea del archivo (MD simul etc)
  fgets(line, sizeof(line), knot); // Segunda linea del archivo (num partic. )
  int npartic = atoi(line); // Número de partículas en la segunda linea 
  fclose(knot);
    if (global_IC_type == 'k') { global_N = npartic; }  
  return global_N; // Devuelve global_N para uso posterior...
} 

////////////////////////// print parameters ////////////////////////////
void print_parameters(struct sim_params *sp, FILE *f)
{  
  printf("\n\n\n");
  printf("         PARAMETERS:\n");
  printf("\n%10s", "");
  printf(" T = %06.3f\n         eta = %06.3f\n           N = %04d\n      f_pull = %06.3f\n",sp->T,sp->eta,sp->N,sp->f_pull);
  printf("%4s", "");
  printf(" IC_type = %c\n   meas_freq = %06.3f\n    sim_time = %08.1f\nfile_to_load = %s\n",sp->IC_type,sp->meas_freq,sp->sim_time,sp->file_to_load);
} 

// Función para limpiar la pantalla
void limpiar_pantalla() {
    system("clear"); // Ejecuta el comando "clear" en sistemas basados en Unix
}

// Funciones
void cursor_a_1_1() { printf("\033[1;1H"); }     // Cursor a fila 1, columna 1
void cursor_a_16_20() { printf("\033[16;20H"); } // Cursor a fila 16, columna 20
void cursor_a_17_20() { printf("\033[17;20H"); } // Cursor a (fila, columna)
void cursor_a_16_40() { printf("\033[16;40H"); } // Cursor a (fila, columna)
void cursor_a_17_40() { printf("\033[17;40H"); } //     ""
void cursor_a_16_60() { printf("\033[16;60H"); } //     ""
void cursor_a_17_60() { printf("\033[17;60H"); } //     ""
void cursor_a_16_80() { printf("\033[16;80H"); } //     ""
void cursor_a_17_80() { printf("\033[17;80H"); } //     ""
void cursor_a_17_98() { printf("\033[20;40H"); } // Cursor a fila 17, columna 98
void cursor_a_20_40() { printf("\033[20;40H"); } // Cursor a fila 20, columna 40
void cursor_a_22_20() { printf("\033[22;20H"); } // Cursor a fila 22, columna 20

void estado_input_x() {
    if (tecla == '1') {
      input_in_x = 1;
      input_in_y = 0;
      input_in_z = 0;
      input_in_n_atom = 0;
    }
}    
void estado_input_y() {
    if (tecla == '2') {
      input_in_x = 0;
      input_in_y = 1;
      input_in_z = 0;
      input_in_n_atom = 0;
    }
}
void estado_input_z() {
    if (tecla == '3') {
      input_in_x = 0;
      input_in_y = 0;
      input_in_z = 1;
      input_in_n_atom = 0;
    }
}
void estado_input_n_atom() {
    if (tecla == '4') {
      input_in_x = 0;
      input_in_y = 0;
      input_in_z = 0;
      input_in_n_atom = 1;
    }
}
////////////////////////////  PRESENTACION DE INICIO  //////////////////////////////////

void presentacion_inicio() 
{
    printf("///////////////////////////////////////////////////////////////////////////////////////////\n");
    printf("//                                                                                       //\n");
    printf("//                                 PARA QUE QUEDE CLARO:                                 //\n");
    printf("//                                                                                       //\n");
    printf("//        ESTE PROGRAMA GENERA UN FICHERO DE POSICIONES INICIALES DE LOS MONOMEROS       //\n");
    printf("//        CON LOS QUE LUEGO ALIMENTAREMOS AL PROGRAMA simulate-polimer.bin               //\n");   
    printf("//        INSTRUCCIONES: moveremos la posición de cada partícula mediante el teclado     //\n");
    printf("//        VER INSTRUCCIONES PULSANDO LA TECLA ENTER                                      //\n");
    printf("//                                                                                       //\n");
    printf("///////////////////////////////////////////////////////////////////////////////////////////\n\n");
 
    printf("Este texto de encima será borrado al pulsar enter.\n");
   
    getchar();
    // Limpia la pantalla y muestra las posiciones iniciales
    printf("\033[2J"); // Limpiar pantalla
    cursor_a_1_1(); // IR A ORIGEN
}
void instrucciones() 
{
    printf("             /////////////////////////////////////////////////////////////////////////////////////////////////////\n");
    printf("             //                                                                                                 //\n");
    printf("             //                                 INSTRUCCIONES:                                                  //\n");
    printf("             //     PARA MODIFICAR X PULSAREMOS 1 Y DESPUES ENTER, DESPUES LA TECLA + O LA - Y DESPUES ENTER    //\n");
    printf("             //     PARA MODIFICAR Y PULSAREMOS 2 Y DESPUES ENTER, DESPUES LA TECLA + O LA - Y DESPUES ENTER    //\n");
    printf("             //     PARA MODIFICAR Z PULSAREMOS 3 Y DESPUES ENTER, DESPUES LA TECLA + O LA - Y DESPUES ENTER    //\n");
    printf("             //     PARA MODIFICAR n PULSAREMOS 4 Y DESPUES ENTER, DESPUES LA TECLA + O LA - Y DESPUES ENTER    //\n");
    printf("             //        n es el número de la partícula que vamos a almacenar en el fichero de nudo               //\n");
    printf("             //        PARA VALIDAR LA POSICION DE LA PARTICULA PULSAR 9 Y DESPUES ENTER                        //\n");
    printf("             //        PARA SALIR DEL PROGRAMA PULSAR q Y DESPUES ENTER                                         //\n");
    printf("             //  NOTA: para generar completo el archivo de nudo hay que meter antes en el directorio            //\n");
    printf("             //  de knots-generator.bin un fichero de texto (vacio) con el nombre del nudo. Este                //\n");
    printf("             //  nombre hay que cambiarlo antes de lanzar el generador de nudos en el fichero parameters.dat    //\n");
    printf("             /////////////////////////////////////////////////////////////////////////////////////////////////////\n");
}
void linea_estados_de_entrada_de_datos() {
      // Indicar la posición del dato a meter dinámicamente
      cursor_a_16_20();
      if (input_in_x == 1) { printf("↑ con +/↓ con -                                                                 "); }
      cursor_a_16_20();
      if (input_in_y == 1) { printf("                    ↑ con +/↓ con -                                             "); }
      cursor_a_16_20();
      if (input_in_z == 1) { printf("                                        ↑ con +/↓ con -                          "); }
      cursor_a_16_20();
      if (input_in_n_atom == 1) { printf("                                                           ↑ con +/↓ con -"); }
      
      cursor_a_20_40();  // Después de imprimir cursor a posición fija.
      }

void linea_con_xyz_y_n_atom() {
      // Mostrar vector posición y número de átomo
      cursor_a_17_20();
      printf(" Pos X = %.3f", x);
      cursor_a_17_40();
      printf(" Pos Y = %.3f", y);
      cursor_a_17_60();
      printf(" Pos Z = %.3f", z);
      cursor_a_17_80();
      printf(" no atom = %d  ", n_atom);
      
      cursor_a_20_40();  // Después de imprimir cursor a posición fija.
}
void analiza_tecla()  {
      // Comprobar si la tecla es '+', '-',
      if (tecla == '+') { bajar = 0; subir = 1;}
      if (tecla == '-') { bajar = 1; subir = 0;}
      // Comprobar si la tecla es '9'
      if (tecla == '9') { atomo_validado = 1;}
}
void gosub_actualizar_x() {
      if (input_in_x == 1 && subir == 1) { x = x + 1; }
      if (input_in_x == 1 && bajar == 1) { x = x - 1; }
}
void gosub_actualizar_y() {
      if (input_in_y == 1 && subir == 1) { y = y + 1; }
      if (input_in_y == 1 && bajar == 1) { y = y - 1; }
}
void gosub_actualizar_z() {
      if (input_in_z == 1 && subir == 1) { z = z + 1; }
      if (input_in_z == 1 && bajar == 1) { z = z - 1; }
}
void gosub_actualizar_n_atom() {
      if (input_in_n_atom == 1 && subir == 1) { n_atom = n_atom + 1; }
      if (input_in_n_atom == 1 && bajar == 1) { n_atom = n_atom - 1; }
}
///////////////////  ESCRITURA EN FICHERO DE LAS POSICIONES (GPT) /////////
void gosub_write_particle_to_file(const char *file_copy, int n_atom, float x, float y, float z) { 

    FILE *file = fopen(file_copy, "a+");
    if (file == NULL) {
        perror("Error al abrir el archivo");
        return;
    }
    // Escribir la partícula en el archivo con el formato especificado
    fprintf(file, "    %dX        X    %d  %7.3f  %7.3f  %7.3f\n",
            n_atom, n_atom, x, y, z);

    fclose(file);
}

//////////////////////////    MAIN     //////////////////////////////     

////////// OTRA SECCION MAS COPIADA DE MAIN DE simulate-polimer.cu /////////// 

int main( int argc, char const *argv[])
{
    // ================== DIRECTORIO =====================
    /// argv[0]: nombre del programa y argv[1]: directorio de simulación (sim_dir) 
    if( argc!=2){  //si el número de argumentos es distinto de 2
      if( argc<2){ printf("You forgot the input.\n"); exit(-1);}//si es menor de 2
      else{ printf("Too many arguments.\n"); exit(-1);} // si mayor de 2 => exit
    }
    if( sizeof(argv[1])>128){ printf("Directory name too long.\n"); exit(-1);}
    char sim_dir[128];
    snprintf(sim_dir,sizeof(sim_dir),"%s",argv[1]);

    // Simulation parameters and variables ////////////////////////////////////

    struct sim_params sp;
      FILE *param_file; // Declaramos el puntero a archivo
      param_file = fopen("/home/marcomc/Documentos/Program/Simulations/Test4/parameters.dat", "r");
      // Abrimos el archivo en modo lectura ("r") y asignamos el puntero
      if (param_file == NULL) {
          perror(" Error al abrir parameters.dat");
          return 1;
      }
    // Leer el archivo DEL DISCO: parameters.dat y meterlos en param_file
    read_parameters(&sp, param_file); // se leen los parámetros 
    print_parameters(&sp, stdout); // se imprimen los parámetros en pantalla
    fclose(param_file); // cerramos el archivo. 
  
    make_global_N(); // make_global_N hacia linea 90
    printf("global_N = number of particles = %d\n", global_N);
    /////////////// FIN COPIA de simulate polimer.cu  /////////////

    presentacion_inicio(); // Gran pantallazo paquetenteres!

    instrucciones();  // Otro gran pantallazo paquetenteres!
      
    linea_estados_de_entrada_de_datos();  // Mostrar la posición del dato a meter
    
    linea_con_xyz_y_n_atom();  // Mostrar vector posición y número de átomo
    
    while (1) {
        
        tecla = getchar();     // Leer la tecla presionada

        // Si el carácter actual es '\n', el anterior está en tecla_ant (bucle)      
        if (tecla == '\n') { tecla_ant = tecla; }
                   
        estado_input_x();      // bit de estado: genera input_in_x con tecla=1
        estado_input_y();      // bit de estado: genera input_in_y
        estado_input_z();      // bit de estado: genera input_in_z
        estado_input_n_atom(); // bit de estado: genera input_in_n_atom

        analiza_tecla();  // Controla si pulsamos + ó - y cambia subir/bajar
             
        fflush(stdout); // Asegurarse de que todo se imprima correctamente
                                                      
        if (subir == 1 || bajar == 1) {
        gosub_actualizar_x(); // Adivina: actualiza x 
        gosub_actualizar_y(); // Actualiza y ademas de 
        gosub_actualizar_z(); // Actualiza z ademas de 
        gosub_actualizar_n_atom(); // Actualiza n_atom
        //                                                          linea_estados_de_entrada_de_datos();
        //                                                          linea_con_xyz_y_n_atom();
        subir = 0;
        bajar = 0;
        cursor_a_20_40();
        }
        linea_estados_de_entrada_de_datos();
        linea_con_xyz_y_n_atom();
        
        if (atomo_validado == 1) {
        gosub_write_particle_to_file(file_copy, n_atom, x, y, z);
        cursor_a_17_98(); 
        printf("atomo_validado = %d", n_atom);
        fflush(stdout); // Asegurarse de que todo se imprima correctamente
        atomo_validado = 0;
        cursor_a_20_40();
        }

        fflush(stdout); // Asegurarse de que el cambio se vea de inmediato
        cursor_a_22_20();
        printf("Recuerde: x: tecla 1, y: tecla 2, z:tecla 3, n atomo: 4 y validar atomo con tecla 9");
        fflush(stdout); 
        cursor_a_20_40();  // Cursor: te he dicho que a (13,40).

        // Salir si se presiona 'q'
        if (tecla == 'q') {
            printf("Saliendo del programa...");
            break;
        }
        // Limpiamos el buffer de teclado.
        while (getchar() != '\n'); // Limpiar cualquier entrada sobrante
        cursor_a_20_40();  // Cursor: te he dicho que a (13,40).
    }

    return 0;
}

